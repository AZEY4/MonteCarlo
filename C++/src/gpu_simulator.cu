#include "hip/hip_runtime.h"
#include "gpu_simulator.cuh"
#include "option.hpp"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <vector>
#include <cstring>
#include <cstdint>
#include <cstdio>

// This file implements two kernels:
//  - mc_kernel_european: pathless European payoff.
//  - mc_kernel_asian: path-based Asian option (discrete arithmetic average).

// European kernel (pathless)
__global__ void mc_kernel_european(Option opt, std::size_t num_paths, double *d_payoffs, unsigned long long seed) {
    std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    // initialize RNG per-thread
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed, idx, 0, &state);

    double z = hiprand_normal_double(&state);
    double ST = opt.S0 * exp((opt.r - 0.5 * opt.sigma * opt.sigma) * opt.T + opt.sigma * sqrt(opt.T) * z);
    double payoff = 0.0;
    if (opt.type == OptionType::EuropeanCall) payoff = fmax(ST - opt.K, 0.0);
    else if (opt.type == OptionType::EuropeanPut) payoff = fmax(opt.K - ST, 0.0);
    d_payoffs[idx] = payoff;
}

// Asian kernel (discrete arithmetic average)
__global__ void mc_kernel_asian(Option opt, std::size_t num_paths, int steps, double *d_payoffs, unsigned long long seed) {
    std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    hiprandStatePhilox4_32_10_t state;
    // Use idx as subsequence to ensure different sequences per thread
    hiprand_init(seed, idx, 0, &state);

    double dt = opt.T / static_cast<double>(steps);
    double drift_dt = (opt.r - 0.5 * opt.sigma * opt.sigma) * dt;
    double vol_sqrt_dt = opt.sigma * sqrt(dt);

    double S = opt.S0;
    double avg = 0.0;
    // simulate discrete path
    for (int t = 0; t < steps; ++t) {
        double z = hiprand_normal_double(&state);
        S *= exp(drift_dt + vol_sqrt_dt * z);
        avg += S;
    }
    avg /= static_cast<double>(steps);

    double payoff = 0.0;
    if (opt.type == OptionType::AsianCall) payoff = fmax(avg - opt.K, 0.0);
    else if (opt.type == OptionType::AsianPut) payoff = fmax(opt.K - avg, 0.0);

    d_payoffs[idx] = payoff;
}

// Host wrapper
extern "C" void monte_carlo_gpu_c(const Option *opt_ptr, std::size_t num_paths, double *out_price, double *out_stderr) {
    if (!opt_ptr || !out_price || !out_stderr) return;
    Option opt = *opt_ptr; // copy to local

    // basic validation
    if (num_paths == 0) {
        *out_price = 0.0;
        *out_stderr = 0.0;
        return;
    }

    // allocate device memory
    double *d_payoffs = nullptr;
    size_t bytes = num_paths * sizeof(double);
    hipError_t err = hipMalloc((void**)&d_payoffs, bytes);
    if (err != hipSuccess) {
        // allocation failed
        *out_price = NAN; *out_stderr = NAN;
        return;
    }

    const int block = 256;
    int grid = static_cast<int>((num_paths + block - 1) / block);
    unsigned long long seed = 123456789ULL;

    // Dispatch kernel based on option type
    if (opt.type == OptionType::EuropeanCall || opt.type == OptionType::EuropeanPut) {
        mc_kernel_european<<<grid, block>>>(opt, num_paths, d_payoffs, seed);
    } else {
        // Asian option: choose steps.
        const int steps = 100;
        mc_kernel_asian<<<grid, block>>>(opt, num_paths, steps, d_payoffs, seed);
    }

    // sync and check
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        hipFree(d_payoffs);
        *out_price = NAN; *out_stderr = NAN;
        return;
    }

    // copy back to host
    std::vector<double> h_payoffs;
    try {
        h_payoffs.resize(num_paths);
    } catch (...) {
        hipFree(d_payoffs);
        *out_price = NAN; *out_stderr = NAN;
        return;
    }
    err = hipMemcpy(h_payoffs.data(), d_payoffs, bytes, hipMemcpyDeviceToHost);
    hipFree(d_payoffs);
    if (err != hipSuccess) {
        *out_price = NAN; *out_stderr = NAN;
        return;
    }

    // host-side reduction to compute mean and stderr
    double sum = 0.0;
    for (std::size_t i = 0; i < num_paths; ++i) sum += h_payoffs[i];
    double mean = sum / static_cast<double>(num_paths);

    double sq_sum = 0.0;
    for (std::size_t i = 0; i < num_paths; ++i) {
        double d = h_payoffs[i] - mean;
        sq_sum += d * d;
    }
    double sample_var = (num_paths > 1) ? (sq_sum / static_cast<double>(num_paths - 1)) : 0.0;
    double stderr = (num_paths > 0) ? sqrt(sample_var / static_cast<double>(num_paths)) : 0.0;

    *out_price = exp(-opt.r * opt.T) * mean;
    *out_stderr = exp(-opt.r * opt.T) * stderr;
}
